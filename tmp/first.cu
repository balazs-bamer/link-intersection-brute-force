#include "hip/hip_runtime.h"
/*
 * first.cu
 *
 *  Created on: 2021 jan. 10
 *      Author: balazs
 */

#include "stl_reader.h"
#include <array>
#include <deque>
#include <chrono>
#include <random>
#include <vector>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <algorithm>
#include <Eigen/Dense>


using Triangle = std::array<Eigen::Vector3f, 3u>;
using CudaTriangle = Eigen::Vector3f*;
using CudaConstTriangle = Eigen::Vector3f const*;
using TrianglesDeque = std::deque<Triangle>;
using TrianglesVector = std::vector<Eigen::Vector3f>;
using Result = std::vector<int8_t>;

constexpr float   cgEpsilon        = 0.00001f;       // TODO consider if uniform epsilon suits all needs.
constexpr uint32_t cgSignumZero     = 0u;
constexpr uint32_t cgSignumPlus     = 1u;
constexpr uint32_t cgSignumMinus    = 2u;
constexpr uint32_t cgSignumShift0   = 0u;
constexpr uint32_t cgSignumShift1   = 2u;
constexpr uint32_t cgSignumShift2   = 4u;
constexpr uint32_t cgSignumAllZero  = (cgSignumZero  << cgSignumShift0) | (cgSignumZero  << cgSignumShift1) | (cgSignumZero  << cgSignumShift2);
constexpr uint32_t cgSignumAllPlus  = (cgSignumPlus  << cgSignumShift0) | (cgSignumPlus  << cgSignumShift1) | (cgSignumPlus  << cgSignumShift2);
constexpr uint32_t cgSignumAllMinus = (cgSignumMinus << cgSignumShift0) | (cgSignumMinus << cgSignumShift1) | (cgSignumMinus << cgSignumShift2);

constexpr uint32_t cgSignumSelect0a = (cgSignumPlus  << cgSignumShift0) | (cgSignumMinus << cgSignumShift1) | (cgSignumMinus << cgSignumShift2);
constexpr uint32_t cgSignumSelect0b = (cgSignumMinus << cgSignumShift0) | (cgSignumPlus  << cgSignumShift1) | (cgSignumPlus  << cgSignumShift2);
constexpr uint32_t cgSignumSelect0c = (cgSignumZero  << cgSignumShift0) | (cgSignumPlus  << cgSignumShift1) | (cgSignumPlus  << cgSignumShift2);
constexpr uint32_t cgSignumSelect0d = (cgSignumZero  << cgSignumShift0) | (cgSignumMinus << cgSignumShift1) | (cgSignumMinus << cgSignumShift2);
constexpr uint32_t cgSignumSelect0e = (cgSignumPlus  << cgSignumShift0) | (cgSignumZero  << cgSignumShift1) | (cgSignumZero  << cgSignumShift2);
constexpr uint32_t cgSignumSelect0f = (cgSignumMinus << cgSignumShift0) | (cgSignumZero  << cgSignumShift1) | (cgSignumZero  << cgSignumShift2);
constexpr uint32_t cgSignumSelect0g = (cgSignumZero  << cgSignumShift0) | (cgSignumPlus  << cgSignumShift1) | (cgSignumMinus << cgSignumShift2);
constexpr uint32_t cgSignumSelect0h = (cgSignumZero  << cgSignumShift0) | (cgSignumMinus << cgSignumShift1) | (cgSignumPlus  << cgSignumShift2);

constexpr uint32_t cgSignumSelect1a = (cgSignumPlus  << cgSignumShift1) | (cgSignumMinus << cgSignumShift0) | (cgSignumMinus << cgSignumShift2);
constexpr uint32_t cgSignumSelect1b = (cgSignumMinus << cgSignumShift1) | (cgSignumPlus  << cgSignumShift0) | (cgSignumPlus  << cgSignumShift2);
constexpr uint32_t cgSignumSelect1c = (cgSignumZero  << cgSignumShift1) | (cgSignumPlus  << cgSignumShift0) | (cgSignumPlus  << cgSignumShift2);
constexpr uint32_t cgSignumSelect1d = (cgSignumZero  << cgSignumShift1) | (cgSignumMinus << cgSignumShift0) | (cgSignumMinus << cgSignumShift2);
constexpr uint32_t cgSignumSelect1e = (cgSignumPlus  << cgSignumShift1) | (cgSignumZero  << cgSignumShift0) | (cgSignumZero  << cgSignumShift2);
constexpr uint32_t cgSignumSelect1f = (cgSignumMinus << cgSignumShift1) | (cgSignumZero  << cgSignumShift0) | (cgSignumZero  << cgSignumShift2);
constexpr uint32_t cgSignumSelect1g = (cgSignumZero  << cgSignumShift1) | (cgSignumPlus  << cgSignumShift0) | (cgSignumMinus << cgSignumShift2);
constexpr uint32_t cgSignumSelect1h = (cgSignumZero  << cgSignumShift1) | (cgSignumMinus << cgSignumShift0) | (cgSignumPlus  << cgSignumShift2);

// Otherwise select 2, no need for checking and thus no constants.

constexpr uint32_t cgSignumCircumferenceA = (cgSignumZero << cgSignumShift0) | (cgSignumPlus << cgSignumShift1) | (cgSignumPlus << cgSignumShift2);
constexpr uint32_t cgSignumCircumferenceB = (cgSignumZero << cgSignumShift0) | (cgSignumZero << cgSignumShift1) | (cgSignumPlus << cgSignumShift2);
constexpr uint32_t cgSignumCircumferenceC = (cgSignumPlus << cgSignumShift0) | (cgSignumZero << cgSignumShift1) | (cgSignumPlus << cgSignumShift2);
constexpr uint32_t cgSignumCircumferenceD = (cgSignumPlus << cgSignumShift0) | (cgSignumZero << cgSignumShift1) | (cgSignumZero << cgSignumShift2);
constexpr uint32_t cgSignumCircumferenceE = (cgSignumPlus << cgSignumShift0) | (cgSignumPlus << cgSignumShift1) | (cgSignumZero << cgSignumShift2);
constexpr uint32_t cgSignumCircumferenceF = (cgSignumZero << cgSignumShift0) | (cgSignumPlus << cgSignumShift1) | (cgSignumZero << cgSignumShift2);

__device__
constexpr uint32_t calculateSignum(float const distances[3]) {
  uint32_t result = 0u;
  for(int32_t i = 0; i < 3; ++i) {
    int32_t tmp = cgSignumZero;
    if(distances[i] > cgEpsilon) {
      tmp = cgSignumPlus;
    }
    else if(distances[i] < -cgEpsilon) {
      tmp = cgSignumMinus;
    }
    else { // nothing to do
    }
    result |= tmp << (cgSignumShift1 * i);
  }
  return result;
}

__device__
void calculateNormals(Eigen::Vector2f const aShape[3], Eigen::Vector2f aNormals[3]) {
  Eigen::Vector2f side = aShape[1] - aShape[0];
  aNormals[0](0) = -side(1);
  aNormals[0](1) = side(0);
  float correction = 1.0f;
  if(aNormals[0].dot(aShape[2]) < 0.0f) {
    correction = -1.0f;                     // They shall point towards the interior.
    aNormals[0] *= correction;
  }
  else { // nothing to do
  }
  side = aShape[2] - aShape[1];
  aNormals[1](0) = -side(1) * correction;
  aNormals[1](1) = side(0) * correction;
  side = aShape[0] - aShape[2];
  aNormals[2](0) = -side(1) * correction;
  aNormals[2](1) = side(0) * correction;
}

__device__
bool doesTouchOther(uint32_t const aSignums) noexcept {
  bool result = (aSignums == cgSignumAllPlus
  || aSignums == cgSignumCircumferenceA
  || aSignums == cgSignumCircumferenceB
  || aSignums == cgSignumCircumferenceC
  || aSignums == cgSignumCircumferenceD
  || aSignums == cgSignumCircumferenceE
  || aSignums == cgSignumCircumferenceF);
  return result;
}

__device__
bool checkCornerOnPerimeterAndInterior(Eigen::Vector2f aShape1[3], Eigen::Vector2f aShape2[3]) {
  Eigen::Vector2f normals1[3]; // i : i->(i+1)%3
  Eigen::Vector2f normals2[3];
  calculateNormals(aShape1, normals1); // Normal vectors point to the center.
  calculateNormals(aShape2, normals2);
  bool result = false;
  for(int32_t indexCorner = 0; indexCorner < 3; ++indexCorner) {
    float distancesCornerFromEachSideOfShape1[3];
    float distancesCornerFromEachSideOfShape2[3];
    for(int32_t indexSide = 0; indexSide < 3; ++indexSide) {
      distancesCornerFromEachSideOfShape2[indexSide] = normals2[indexSide].dot(aShape1[indexCorner] - aShape2[indexSide]);
      distancesCornerFromEachSideOfShape1[indexSide] = normals1[indexSide].dot(aShape2[indexCorner] - aShape1[indexSide]);
    }
    result = result || doesTouchOther(calculateSignum(distancesCornerFromEachSideOfShape2));
    result = result || doesTouchOther(calculateSignum(distancesCornerFromEachSideOfShape1));   // True if one corner is on the sides, corners or interior of the other triangle.
    // Don't break out since it won't use on CUDA.
  }
  return result;
}

__device__
bool checkTrueIntersecitonOfSides(Eigen::Vector2f aShape1[3], Eigen::Vector2f aShape2[3]) {
  bool result = false;
  for(int32_t indexSide1 = 0; indexSide1 < 3; ++indexSide1) {
    for(int32_t indexSide2 = 0; indexSide2 < 3; ++indexSide2) {
      Eigen::Vector2f &side1a = aShape1[indexSide1];
      Eigen::Vector2f &side1b = aShape1[(indexSide1 + 1) % 3];
      Eigen::Vector2f &side2a = aShape2[indexSide2];
      Eigen::Vector2f &side2b = aShape2[(indexSide2 + 1) % 3];
      // Manually solve linear EQ to make sure we have as few branches as possible.
      uint32_t nonzeroAindex = (fabs(side1a(0) - side1b(0)) > cgEpsilon ? 0 : 1);
      float a = side1a(nonzeroAindex) - side1b(nonzeroAindex);
      float b = side2b(nonzeroAindex) - side2a(nonzeroAindex);
      float c = side1a(1 - nonzeroAindex) - side1b(1 - nonzeroAindex);
      float d = side2b(1 - nonzeroAindex) - side2a(1 - nonzeroAindex);
      float determinant = a * d - b * c;
      if(fabs(determinant) > cgEpsilon) {
        float k = side1a(nonzeroAindex) - side2a(nonzeroAindex);
        float l = side1a(1 - nonzeroAindex) - side2a(1 - nonzeroAindex);
        float v = (l * a - c * k) / determinant;
        float u = (k - b * v) / a;
        result = result || (u >= 0.0f && u <= 1.0f && v >= 0.0f && v <= 1.0f);  // The intersection is inside of both sides.
      }
      else { // nothing to do, because the lines are parallel but can't touch each other.
      }
    }
  }
  return result;
}

__device__
bool hasCommonPoint(CudaConstTriangle const aShape1, CudaConstTriangle const aShape2, Eigen::Vector3f const &aShape1normal, Eigen::Vector3f const &aShape2normal) { // coplanar
  Eigen::Vector3f normal;
  if(aShape1normal.dot(aShape2normal) > 0.0f) {
    normal = aShape1normal + aShape2normal;
  }
  else {
    normal = aShape1normal - aShape2normal;
  }
  int32_t indexX = 1;
  int32_t indexY = 2;
  float abs1 = fabs(normal(1)); // Looking for the biggest projection. TODO fabsf for CUDA
  if(abs1 > fabs(normal(0))) {
    indexX = 0;
  }
  else { // nothing to do
  }
  if(fabs(normal(2)) > abs1) {
    indexX = 0;
    indexY = 1;
  }
  else { // nothing to do
  }
  Eigen::Vector2f shape1[3];
  Eigen::Vector2f shape2[3];
  for(int32_t i = 0; i < 3; ++i) {
    shape1[i](0) = aShape1[i](indexX);  // Project 3D triangle to axis-parallel plane.
    shape1[i](1) = aShape1[i](indexY);
    shape2[i](0) = aShape2[i](indexX);
    shape2[i](1) = aShape2[i](indexY);
  }
  bool result = checkCornerOnPerimeterAndInterior(shape1, shape2);
  if(!result) { // Common point may only occur now when sides truly intersect each other.
    result = checkTrueIntersecitonOfSides(shape1, shape2);
  }
  else { // nothing to do
  }
  return result;
}

__device__
void calculateIntersectionParameter(
  CudaConstTriangle const aShape
, Eigen::Vector3f const &aIntersectionVector
, float const aDistanceCornerNfromOtherPlane[3]
, uint32_t const aSignumShapeFromOtherPlane
, float &aIntersectionParameterA
, float &aIntersectionParameterB) noexcept {
  int32_t indexCommon, indexA, indexB;
  if(aSignumShapeFromOtherPlane == cgSignumSelect0a
  || aSignumShapeFromOtherPlane == cgSignumSelect0b
  || aSignumShapeFromOtherPlane == cgSignumSelect0c
  || aSignumShapeFromOtherPlane == cgSignumSelect0d
  || aSignumShapeFromOtherPlane == cgSignumSelect0e
  || aSignumShapeFromOtherPlane == cgSignumSelect0f
  || aSignumShapeFromOtherPlane == cgSignumSelect0g
  || aSignumShapeFromOtherPlane == cgSignumSelect0h) {
    indexCommon = 0; indexA = 1; indexB = 2;
  }
  else if(aSignumShapeFromOtherPlane == cgSignumSelect1a
  || aSignumShapeFromOtherPlane == cgSignumSelect1b
  || aSignumShapeFromOtherPlane == cgSignumSelect1c
  || aSignumShapeFromOtherPlane == cgSignumSelect1d
  || aSignumShapeFromOtherPlane == cgSignumSelect1e
  || aSignumShapeFromOtherPlane == cgSignumSelect1f
  || aSignumShapeFromOtherPlane == cgSignumSelect1g
  || aSignumShapeFromOtherPlane == cgSignumSelect1h) {
    indexCommon = 1; indexA = 0; indexB = 2;
  }
  else {
    indexCommon = 2; indexA = 1; indexB = 0;
  }
  float vertexProjections[3];
  for(int32_t i = 0; i < 3; ++i) {
    vertexProjections[i] = aIntersectionVector.dot(aShape[i]);
  }
  aIntersectionParameterA =
   vertexProjections[indexA]
 + (vertexProjections[indexCommon] - vertexProjections[indexA])
 * aDistanceCornerNfromOtherPlane[indexA]
 / (aDistanceCornerNfromOtherPlane[indexA] - aDistanceCornerNfromOtherPlane[indexCommon]);
  aIntersectionParameterB =
   vertexProjections[indexB]
 + (vertexProjections[indexCommon] - vertexProjections[indexB])
 * aDistanceCornerNfromOtherPlane[indexB]
 / (aDistanceCornerNfromOtherPlane[indexB] - aDistanceCornerNfromOtherPlane[indexCommon]);
}

__global__
void hasCommonPoint(int32_t aPairCount, CudaConstTriangle const aShapes, int32_t const * const aIndices, int8_t * const aResult) {
  int32_t indexThread = blockIdx.x*blockDim.x + threadIdx.x;
  if(indexThread < aPairCount) {
    bool result = false;
    CudaConstTriangle shape1 = aShapes + aIndices[indexThread * 2] * 3;
    CudaConstTriangle shape2 = aShapes + aIndices[indexThread * 2 + 1] * 3;
    Eigen::Vector3f shape1normal = (shape1[1] - shape1[0]).cross(shape1[2] - shape1[0]);
    Eigen::Vector3f shape2normal = (shape2[1] - shape2[0]).cross(shape2[2] - shape2[0]);
    shape1normal.normalize();
    shape2normal.normalize();
    float distanceCornerNofShape1FromPlane2[3];
    float distanceCornerNofShape2FromPlane1[3];
    for(int32_t i = 0; i < 3; ++i) {
      distanceCornerNofShape1FromPlane2[i] = shape2normal.dot(shape1[i] - shape2[0]);
      distanceCornerNofShape2FromPlane1[i] = shape1normal.dot(shape2[i] - shape1[0]);
    }
    uint32_t signumShape1FromPlane2 = calculateSignum(distanceCornerNofShape1FromPlane2); // These contain info about relation of each point and the other plane.
    uint32_t signumShape2FromPlane1 = calculateSignum(distanceCornerNofShape2FromPlane1);
    if(signumShape1FromPlane2 == cgSignumAllPlus || signumShape1FromPlane2 == cgSignumAllMinus || signumShape2FromPlane1 == cgSignumAllPlus || signumShape2FromPlane1 == cgSignumAllMinus) {
      // Nothing to do: one triangle is completely on the one side of the other's plane
    }
    else {
      Eigen::Vector3f intersectionVector = shape1normal.cross(shape2normal);
      if(intersectionVector.norm() > cgEpsilon && signumShape1FromPlane2 != cgSignumAllZero && signumShape2FromPlane1 != cgSignumAllZero) { // Real intersection, planes are not identical, and both triangles touch the common line.
        intersectionVector.normalize();
        float intersectionParameterAshape1;
        float intersectionParameterBshape1;
        float intersectionParameterAshape2;
        float intersectionParameterBshape2;
        calculateIntersectionParameter(shape1, intersectionVector, distanceCornerNofShape1FromPlane2, signumShape1FromPlane2, intersectionParameterAshape1, intersectionParameterBshape1); // The two parameters will contain the locations of the touching point.
        calculateIntersectionParameter(shape2, intersectionVector, distanceCornerNofShape2FromPlane1, signumShape2FromPlane1, intersectionParameterAshape2, intersectionParameterBshape2);
        if(intersectionParameterAshape1 > intersectionParameterBshape1) {
          auto tmp = intersectionParameterAshape1;
          intersectionParameterAshape1 = intersectionParameterBshape1;
          intersectionParameterBshape1 = tmp;
        }
        else { // nothing to do
        }
        if(intersectionParameterAshape2 > intersectionParameterBshape2) {
          auto tmp = intersectionParameterAshape2;
          intersectionParameterAshape2 = intersectionParameterBshape2;
          intersectionParameterBshape2 = tmp;
        }
        else { // nothing to do
        }
        if((intersectionParameterAshape1 - cgEpsilon <= intersectionParameterAshape2) && (intersectionParameterAshape2 <= intersectionParameterBshape1 + cgEpsilon) // Epsilons make possible to check for triangles with corner-corner or corner-edge touch.
        || (intersectionParameterAshape1 - cgEpsilon <= intersectionParameterBshape2) && (intersectionParameterBshape2 <= intersectionParameterBshape1 + cgEpsilon)
        || (intersectionParameterAshape2 - cgEpsilon <= intersectionParameterAshape1) && (intersectionParameterAshape1 <= intersectionParameterBshape2 + cgEpsilon)
        || (intersectionParameterAshape2 - cgEpsilon <= intersectionParameterBshape1) && (intersectionParameterBshape1 <= intersectionParameterBshape2 + cgEpsilon)) {
          result = true;
        }
        else { // nothing to do
        }
      }
      else {
        result = hasCommonPoint(shape1, shape2, shape1normal, shape2normal); // Coplanar triangles
      }
    }
    aResult[indexThread] = (result ? 1 : 0);
  }
  else { // nothing to do
  }
}

Eigen::Matrix3f randomTransform(std::default_random_engine &aGenerator) {
  std::uniform_real_distribution<float> distribution(0.0f, 1.0f);
  Eigen::Matrix3f result;
  for(int32_t i = 0; i < 9; ++i) {
    result(i / 3, i % 3) = distribution(aGenerator);
  }
  return result;
}

TrianglesDeque readTriangles(char const * const aFilename) {
  TrianglesDeque result;
  stl_reader::StlMesh<float, int32_t> mesh(aFilename);
  for(int32_t indexTriangle = 0; indexTriangle < mesh.num_tris(); ++indexTriangle) {
      Triangle triangle;
      for(int32_t indexCorner = 0; indexCorner < 3; ++indexCorner) {
          float const * const coords = mesh.tri_corner_coords(indexTriangle, indexCorner);
          Eigen::Vector3f in;
          for(int32_t i = 0; i < 3; ++i) {
            in(i) = coords[i];
          }
          triangle[indexCorner] = in;
      }
      result.push_back(triangle);
  }
  return result;
}

void writeTriangles(TrianglesDeque const &aTriangles, char const * const aFilename) {
  std::ofstream out(aFilename);
  out << "solid Exported from Blender-2.82 (sub 7)\n";
  for(auto const & triangle : aTriangles) {
//    Eigen::Vector3f normal = (triangle[1] - triangle[0]).cross(aShape2[2] - aShape2[0]);
//  shape1normal.normalize();
    out << "facet normal 0.000000 0.000000 0.000000\nouter loop\n";
    for(auto const & vertex : triangle) {
      out << "vertex " << vertex(0) << ' ' << vertex(1) << ' ' << vertex(2) << '\n';
    }
    out << "endloop\nendfacet\n";
  }
  out << "endsolid Exported from Blender-2.82 (sub 7)\n";
}

void check(TrianglesVector const &aTriangles, Result &aResult) {
  int32_t count = aTriangles.size() / 3;
  int32_t pairCount = count * (count - 1) / 2;
  int32_t bytesTriangles = count * 3 * sizeof(Eigen::Vector3f);
  int32_t bytesIndices = pairCount * 2 * sizeof(int32_t);

  CudaTriangle trianglesCuda;
  hipMalloc(&trianglesCuda, bytesTriangles);
  hipMemcpy(trianglesCuda, aTriangles.data(), bytesTriangles, hipMemcpyHostToDevice);

  std::vector<int32_t> indicesTmp;
  indicesTmp.reserve(pairCount * 2);
  int32_t unified = 0;
  for(int32_t i = 0; i < count; ++i) {
    for(int32_t j = i + 1; j < count; ++j) {
      indicesTmp[unified++] = i;
      indicesTmp[unified++] = j;
    }
  }
  int32_t *indicesCuda;
  hipMalloc(&indicesCuda, bytesIndices);
  hipMemcpy(indicesCuda, indicesTmp.data(), bytesIndices, hipMemcpyHostToDevice);

  int8_t *resultCuda;
  hipMalloc(&resultCuda, pairCount);

  hasCommonPoint<<<(pairCount + 255) / 256, 256>>>(pairCount, trianglesCuda, indicesCuda, resultCuda);

  hipMemcpy(aResult.data(), resultCuda, pairCount, hipMemcpyDeviceToHost);

  hipFree(indicesCuda);
  hipFree(resultCuda);
  hipFree(trianglesCuda);
}

int main(int argc, char **argv) {
  int ret = 0;
  size_t iterations = 1;
  if(argc < 3) {
    std::cerr << "Usage: " << argv[0] << " <filenameIn> <count>\n";
    ret = 1;
  }
  else {
    try {
      iterations = std::stoul(argv[2]);
      if(iterations == 0u) {
        throw std::invalid_argument("Count must be > 0");
      }
      else { // nothing to do
      }
      std::default_random_engine generator;
      generator.seed((std::chrono::high_resolution_clock::now() - std::chrono::high_resolution_clock::time_point::min()).count());
      auto triangles = readTriangles(argv[1]);
      TrianglesVector trianglesTransform;
      trianglesTransform.reserve(triangles.size() * 3);
      Result result;
      auto count = triangles.size();
      auto pairCount = count * (count - 1u) / 2u;
      result.reserve(pairCount);
      std::fill_n(result.begin(), pairCount, 0);

      auto start = std::chrono::high_resolution_clock::now();
      for(size_t i = 0; i < iterations; ++i) {
        auto transform = randomTransform(generator);
        trianglesTransform.clear();
        for(auto &item : triangles) {
          std::transform(item.begin(), item.end(), std::back_inserter(trianglesTransform), [&transform](auto &item) -> Eigen::Vector3f {
            Eigen::Vector3f result = transform * item;
            return result;
          });
        }
        check(trianglesTransform, result);
      }
      auto finish = std::chrono::high_resolution_clock::now();

      size_t unified = 0;
      for(int32_t i = 0; i < count; ++i) {
        for(int32_t j = i + 1; j < count; ++j) {
          if(result[unified]) {
            std::cout << "Has common point: " << i << ' ' << j << "\n";
          }
          else { // nothing to do
          }
          ++unified;
        }
      }
      double totalTime = std::chrono::duration_cast<std::chrono::duration<double>>(finish - start).count();

      std::cout << "Average time: " << totalTime / iterations << '\n';
    }
    catch(std::exception &e) {
      std::cerr << "exception: " << e.what() << '\n';
      ret = 2;
    }
  }
  return ret;
}
